
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <math.h>

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N);

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {

   int ROW = blockIdx.y * blockDim.y + threadIdx.y;
   int COL = blockIdx.x * blockDim.x + threadIdx.x;

   float tmpSum = 0;

   if (ROW < N && COL < N) {
      /* Each thread computes a single element of the block */
      for (int i = 0; i < N; i++) {
         tmpSum += A[ROW * N + i] * B[i * N + COL];
      }
   }
   C[ROW * N + COL] = tmpSum;
}

int main() {
   size_t N = 16;
   /* Matrix dimension */
   size_t SIZE = N * N;

   /* Declare device memory pointers */
   float *d_A;
   float *d_B;
   float *d_C;

   /* Allocate CUDA memory */
   hipMallocManaged(&d_A, SIZE * sizeof(float));
   hipMallocManaged(&d_B, SIZE * sizeof(float));
   hipMallocManaged(&d_C, SIZE * sizeof(float));

   /* Fill values into A and B */
   for (int i { 0 }; i < N; i++) {
      for (int j { 0 }; j < N; j++) {
         d_B[N * i + j] = cos(j);
         d_A[j + N * i] = sin(i);
      }
   }

   /* Define grid and block dimensions */
   dim3 block_dim;
   dim3 grid_dim;

   if (N * N > 512) {
      block_dim = {512, 512};
      grid_dim = {(N + 512 - 1) / 512, (N + 512 - 1) / 512};
   } else {
      block_dim = {N, N};
      grid_dim = {1, 1};
   }

   /* Invoke kernel */
   matrixMultiplicationKernel<<<grid_dim,block_dim>>>(d_A, d_B, d_C, N);

   /* Synchronize device and host memory */
   hipDeviceSynchronize();

   float *cpu_C;
   cpu_C = new float[SIZE];

   /* Run matrix multiplication on the CPU for reference */
   float sum;
   for (int row { 0 }; row < N; row++) {
      for (int col { 0 }; col < N; col++) {
         sum = 0.f;
         for (int n { 0 }; n < N; n++) {
            sum += d_A[row * N + n] * d_B[n * N + col];
         }
         cpu_C[row * N + col] = sum;
      }
   }

   double err { 0 };
   /* Compare the results */
   for (int ROW { 0 }; ROW < N; ROW++) {
      for (int COL { 0 }; COL < N; COL++) {
         err += cpu_C[ROW * N + COL] - d_C[ROW * N + COL];
      }
   }

   std::cout << "Error: " << err << std::endl;

   /* Free the CUDA memory */
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   return 0;
}
